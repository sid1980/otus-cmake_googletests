#include "hip/hip_runtime.h"
// 一维数组相加
//https://blog.csdn.net/xuhang0910/article/details/51941309

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#define N 10

__global__ void add(int *a, int *b, int *c)
{
    int tid =blockIdx.x;
    c[tid] = a[tid] + b[tid];

}
int main()
{
    int a[N], b[N], c[N];
    int *deva, *devb, *devc;
    //在GPU上分配内存
    hipMalloc((void **)&deva, N*sizeof(int));
    hipMalloc((void **)&devb, N*sizeof(int));
    hipMalloc((void **)&devc, N*sizeof(int));

    //在CPU上为数组赋值
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i*i;
    }
    //将数组a和b传到GPU
    hipMemcpy(deva, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devb, b, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devc, c, N*sizeof(int), hipMemcpyHostToDevice);
    add <<<N, 1 >> >(deva, devb, devc);

    //将数组c从GPU传到CPU
    hipMemcpy(c, devc, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
        printf("%d+%d=%d\n", a[i], b[i], c[i]);
    }
    hipFree(deva);
    hipFree(devb);
    hipFree(devc);
    return 0;
}